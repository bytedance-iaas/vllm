#include "hip/hip_runtime.h"
#include <vector>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/group_array_problem_shape.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/mixed_dtype_utils.hpp"

using namespace cute;

// Type definitions
using MmaType = cutlass::float_e4m3_t;      // FP8 e4m3 type
using QuantType = cutlass::int4b_t;         // 4-bit integer type
using ElementAccumulator = float;           // Accumulator type
using ElementScale = float;                 // Scale type
using ElementC = cutlass::half_t;           // Default output type (FP16)
using ElementD = ElementC;                  // Default output type (FP16)
using ProblemShape = cutlass::gemm::GroupProblemShape<Shape<int, int, int>>;

// Architecture-specific configurations
using ArchTag = cutlass::arch::Sm90;
using OperatorClass = cutlass::arch::OpClassTensorOp;
constexpr int TileShapeK = 128 * 8 / sizeof_bits<MmaType>::value;
using TileShape = Shape<_128, _16, cute::Int<TileShapeK>>;
using ClusterShape = Shape<_1, _1, _1>;

// Layout configurations
using LayoutA = cutlass::layout::RowMajor;
using LayoutB = cutlass::layout::ColumnMajor;
using LayoutC = cutlass::layout::RowMajor;
using LayoutD = LayoutC;
using LayoutScale = cutlass::layout::RowMajor;

// Alignments
constexpr int AlignmentA = 128 / cutlass::sizeof_bits<MmaType>::value;
constexpr int AlignmentB = 128 / cutlass::sizeof_bits<QuantType>::value;
constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;

using LayoutA_Transpose = typename cutlass::layout::LayoutTranspose<LayoutA>::type;
using LayoutB_Transpose = typename cutlass::layout::LayoutTranspose<LayoutB>::type;

// Element packing for scales
using ElementScalePacked = cutlass::Array<ElementScale, 1>;

// Kernel schedule and epilogue definitions
using KernelSchedule = cutlass::gemm::KernelPtrArrayTmaWarpSpecializedCooperative;
using EpilogueSchedule = cutlass::epilogue::PtrArrayTmaWarpSpecializedCooperative;

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    TileShape, ClusterShape,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementAccumulator,
    ElementC, typename cutlass::layout::LayoutTranspose<LayoutC>::type *, AlignmentC,
    ElementD, typename cutlass::layout::LayoutTranspose<LayoutD>::type *, AlignmentD,
    EpilogueSchedule
>::CollectiveOp;

using CollectiveMainloopScaleOnly = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    cute::tuple<QuantType, ElementScalePacked>, LayoutB_Transpose *, AlignmentB,
    MmaType, LayoutA_Transpose *, AlignmentA,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    KernelSchedule
>::CollectiveOp;

// Define the final kernel and GEMM operation types
using GemmKernelScaleOnly = cutlass::gemm::kernel::GemmUniversal<
    ProblemShape,
    CollectiveMainloopScaleOnly,
    CollectiveEpilogue
>;

using GemmScaleOnly = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelScaleOnly>;

// Stride definitions
using StrideA = cute::remove_pointer_t<cutlass::detail::TagToStrideA_t<LayoutA*>>;
using StrideB = cute::remove_pointer_t<cutlass::detail::TagToStrideB_t<LayoutB*>>;
using StrideC = typename GemmKernelScaleOnly::InternalStrideC;
using StrideD = typename GemmKernelScaleOnly::InternalStrideD;
using StrideS = typename CollectiveMainloopScaleOnly::StrideScale;

struct Int4Fp8GemmParams {
    // Problem size parameters
    int num_groups;
    typename ProblemShape::UnderlyingProblemShape* problem_sizes; // Sizes of GEMM problems

    // Tensors
    const MmaType **a_ptrs;        // Array of pointers to A matrices
    const QuantType **b_ptrs;      // Array of pointers to B matrices
    const ElementScalePacked **scale_ptrs; // Array of pointers to scale factors
    const ElementC **c_ptrs;       // Array of pointers to C matrices (input)
    ElementD **d_ptrs;             // Array of pointers to D matrices (output)

    // Strides for each tensor
    StrideA *stride_a;        // Strides for A matrices
    StrideB *stride_b;        // Strides for B matrices
    StrideC *stride_c;        // Strides for C matrices
    StrideD *stride_d;        // Strides for D matrices
    const StrideS *stride_s;  // Strides for scales

    // Alpha and beta scaling factors
    ElementAccumulator *alpha;     // Array of alpha values for each problem
    ElementAccumulator *beta;      // Array of beta values for each problem

    // Scale chunk size
    int chunk_size;                // Size of each chunk for scales (typically K/chunks)

    // Workspace memory
    void *workspace;               // Workspace memory
    size_t workspace_size;         // Size of the workspace
};

hipError_t runInt4Fp8GroupedGemm(Int4Fp8GemmParams& params) {
    // Prepare device pointers for alpha and beta if they're not nullptr
    ElementAccumulator** d_alpha_ptr_array = nullptr;
    ElementAccumulator** d_beta_ptr_array = nullptr;

    if (params.alpha && params.beta) {
        hipMalloc(&d_alpha_ptr_array, params.num_groups * sizeof(ElementAccumulator*));
        hipMalloc(&d_beta_ptr_array, params.num_groups * sizeof(ElementAccumulator*));

        // Copy alpha and beta arrays to device
        hipMemcpy(d_alpha_ptr_array, params.alpha, params.num_groups * sizeof(ElementAccumulator*), hipMemcpyHostToDevice);
        hipMemcpy(d_beta_ptr_array, params.beta, params.num_groups * sizeof(ElementAccumulator*), hipMemcpyHostToDevice);
    }

    // Set up GemmUniversalMode and fusion arguments
    using Args = typename GemmScaleOnly::Arguments;
    Args arguments;
    decltype(arguments.epilogue.thread) fusion_args;

    if (params.alpha && params.beta) {
        // Use per-group alpha/beta values
        fusion_args.alpha = 0;
        fusion_args.beta = 0;
        fusion_args.alpha_ptr = nullptr;
        fusion_args.beta_ptr = nullptr;
        fusion_args.alpha_ptr_array = d_alpha_ptr_array;
        fusion_args.beta_ptr_array = d_beta_ptr_array;
        fusion_args.dAlpha = {cute::_0{}, cute::_0{}, 1};
        fusion_args.dBeta = {cute::_0{}, cute::_0{}, 1};
    } else {
        // Use default alpha=1, beta=0
        fusion_args.alpha = 1.0f;
        fusion_args.beta = 0.0f;
        fusion_args.alpha_ptr = nullptr;
        fusion_args.beta_ptr = nullptr;
        fusion_args.alpha_ptr_array = nullptr;
        fusion_args.beta_ptr_array = nullptr;
        fusion_args.dAlpha = {cute::_0{}, cute::_0{}, 0};
        fusion_args.dBeta = {cute::_0{}, cute::_0{}, 0};
    }

    // Create hardware info for the current device
    cutlass::KernelHardwareInfo hw_info;
    hw_info.device_id = 0;
    hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

    // Create gemm arguments
    arguments = Args {
        cutlass::gemm::GemmUniversalMode::kGrouped,
        {params.num_groups, params.problem_sizes, nullptr},
        {params.b_ptrs, params.stride_b, params.a_ptrs, params.stride_a, params.scale_ptrs, params.stride_s, params.chunk_size},
        {fusion_args, params.c_ptrs, params.stride_c, params.d_ptrs, params.stride_d},
        hw_info
    };

    // Instantiate GEMM
    GemmScaleOnly gemm;

    // Get workspace size
    params.workspace_size = GemmScaleOnly::get_workspace_size(arguments);

    // Allocate workspace if not provided
    void* workspace_ptr = params.workspace;
    bool allocated_workspace = false;

    if (!workspace_ptr && params.workspace_size > 0) {
        hipMalloc(&workspace_ptr, params.workspace_size);
        allocated_workspace = true;
    }

    // Check if the problem is supported
    cutlass::Status status = gemm.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
        if (allocated_workspace) {
            hipFree(workspace_ptr);
        }
        if (d_alpha_ptr_array) hipFree(d_alpha_ptr_array);
        if (d_beta_ptr_array) hipFree(d_beta_ptr_array);
        return hipErrorInvalidValue;
    }

    // Initialize the GEMM with arguments and workspace
    status = gemm.initialize(arguments, workspace_ptr);
    if (status != cutlass::Status::kSuccess) {
        if (allocated_workspace) {
            hipFree(workspace_ptr);
        }
        if (d_alpha_ptr_array) hipFree(d_alpha_ptr_array);
        if (d_beta_ptr_array) hipFree(d_beta_ptr_array);
        return hipErrorInvalidValue;
    }

    // Run the GEMM
    status = gemm.run();

    // Free allocated resources
    if (allocated_workspace) {
        hipFree(workspace_ptr);
    }
    if (d_alpha_ptr_array) hipFree(d_alpha_ptr_array);
    if (d_beta_ptr_array) hipFree(d_beta_ptr_array);

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Return status
    return status == cutlass::Status::kSuccess ? hipSuccess : hipErrorUnknown;
}

template <typename StrideType>
std::vector<StrideType> createStrides(const std::vector<torch::Tensor>& tensors, bool is_transposed = false) {
    std::vector<StrideType> strides;
    strides.reserve(tensors.size());

    for (const auto& tensor : tensors) {
        int64_t M = tensor.size(0);
        int64_t N = tensor.size(1);
        if (is_transposed) {
            // For transposed layout
            strides.push_back(cutlass::make_cute_packed_stride(StrideType{}, {static_cast<int>(N), static_cast<int>(M), 1}));
        } else {
            // For standard layout
            strides.push_back(cutlass::make_cute_packed_stride(StrideType{}, {static_cast<int>(M), static_cast<int>(N), 1}));
        }
    }

    return strides;
}

template <typename T>
std::vector<const T*> getDevicePtrs(const std::vector<torch::Tensor>& tensors) {
    std::vector<const T*> ptrs;
    ptrs.reserve(tensors.size());

    for (const auto& tensor : tensors) {
        ptrs.push_back(reinterpret_cast<const T*>(tensor.data_ptr()));
    }

    return ptrs;
}

template <typename T>
std::vector<T*> getMutableDevicePtrs(const std::vector<torch::Tensor>& tensors) {
    std::vector<T*> ptrs;
    ptrs.reserve(tensors.size());

    for (auto& tensor : tensors) {
        // Remove const for output tensors
        ptrs.push_back(reinterpret_cast<T*>(tensor.data_ptr()));
    }

    return ptrs;
}

std::vector<typename ProblemShape::UnderlyingProblemShape> createProblemSizes(
    const std::vector<int>& m_values,
    const std::vector<int>& n_values,
    const std::vector<int>& k_values)
{
    int num_groups = m_values.size();
    std::vector<typename ProblemShape::UnderlyingProblemShape> problem_sizes(num_groups);

    for (int i = 0; i < num_groups; i++) {
        // Note: We swap M and N because of the transpose
        problem_sizes[i] = make_tuple(n_values[i], m_values[i], k_values[i]);
    }

    return problem_sizes;
}

// Helper function to preprocess int4 tensors
std::vector<torch::Tensor> preprocessInt4Tensors(const std::vector<torch::Tensor>& raw_tensors) {
    std::vector<torch::Tensor> processed_tensors;
    processed_tensors.reserve(raw_tensors.size());

    for (const auto& tensor : raw_tensors) {
        // Create a new tensor with the same size to hold the processed values
        auto processed = torch::empty_like(tensor);

        // Call the CUTLASS encoding function
        cutlass::unified_encode_int4b(
            reinterpret_cast<const cutlass::int4b_t*>(tensor.data_ptr()),
            reinterpret_cast<cutlass::int4b_t*>(processed.data_ptr()),
            tensor.numel()
        );

        processed_tensors.push_back(processed);
    }

    return processed_tensors;
}

// Helper function to preprocess scale tensors
std::vector<torch::Tensor> preprocessScaleTensors(const std::vector<torch::Tensor>& scale_tensors) {
    std::vector<torch::Tensor> packed_tensors;
    packed_tensors.reserve(scale_tensors.size());

    for (const auto& tensor : scale_tensors) {
        // Create a tensor for packed scales
        auto packed = torch::empty_like(tensor);

        // Pack the scales using our template function
        cutlass::pack_scale_fp32<ElementScale, ElementScalePacked>(
            reinterpret_cast<const float*>(tensor.data_ptr()),
            reinterpret_cast<ElementScalePacked*>(packed.data_ptr()),
            tensor.numel(),
            ElementScalePacked::kElements
        );

        packed_tensors.push_back(packed);
    }

    return packed_tensors;
}

/**
 * @brief Main function to run int4 * fp8 grouped GEMM from PyTorch
 *
 * This function performs multiple GEMM operations in parallel where each operation multiplies
 * an FP8 matrix (A) with a quantized INT4 matrix (B), applying per-channel scaling factors.
 * It's designed for efficient execution on NVIDIA Hopper GPUs, leveraging Tensor Cores for
 * optimal performance with mixed precision arithmetic.
 *
 * Rather than taking vectors of tensors, this function uses expert_offsets to manage
 * multiple GEMM operations within the unified tensors.
 *
 * @param a_tensor Tensor containing all A matrices (fp8_e4m3) with shape [total_m, K]
 * @param b_tensor Tensor containing all B matrices (int4 packed as int8) with shape [total_n, K/2]
 * @param scale_tensor Tensor containing all scale factors with shape [total_n, K/chunk_size]
 * @param c_tensor Tensor containing all C matrices (input) with shape [total_m, total_n]
 * @param expert_offsets Tensor containing expert offsets for determining group boundaries
 * @param a_strides Optional custom strides for A matrices (nullptr for default)
 * @param b_strides Optional custom strides for B matrices (nullptr for default)
 * @param c_strides Optional custom strides for C matrices (nullptr for default)
 * @param problem_sizes Optional problem sizes (nullptr for auto-detection)
 * @param chunk_size Size of each chunk for scales (K / number of scale chunks); if 0, will be auto-detected
 * @param alpha Optional scalar multiplier for the product of A and B matrices
 * @param beta Optional scalar multiplier for matrix C
 * @return torch::Tensor Output tensor D with shape [total_m, total_n]
 */
torch::Tensor int4Fp8GroupedGemm(
    const torch::Tensor& a_tensor,
    const torch::Tensor& b_tensor,
    const torch::Tensor& scale_tensor,
    const torch::Tensor& c_tensor,
    const torch::Tensor& expert_offsets,
    const StrideA* a_strides = nullptr,
    const StrideB* b_strides = nullptr,
    const StrideC* c_strides = nullptr,
    typename ProblemShape::UnderlyingProblemShape* problem_sizes = nullptr,
    int chunk_size = 0,
    float alpha = 1.0f,
    float beta = 0.0f)
{
    // Check inputs
    TORCH_CHECK(a_tensor.dim() == 2, "A tensor must be 2D");
    TORCH_CHECK(b_tensor.dim() == 2, "B tensor must be 2D");
    TORCH_CHECK(scale_tensor.dim() == 2, "Scale tensor must be 2D");
    TORCH_CHECK(c_tensor.dim() == 2, "C tensor must be 2D");
    TORCH_CHECK(expert_offsets.dim() == 1, "expert_offsets must be a 1D tensor");

    // Get number of groups from expert_offsets
    int num_groups = static_cast<int>(expert_offsets.size(0));

    // Check tensor types
    TORCH_CHECK(a_tensor.scalar_type() == torch::kFloat8_e4m3fn, "A tensor must be fp8 (float_e4m3_t) type");
    TORCH_CHECK(b_tensor.scalar_type() == torch::kInt8, "B tensor must contain packed int4 values (stored as int8)");

    // Set CUDA device based on the input tensor
    const at::cuda::CUDAGuard device_guard(a_tensor.device());

    // Create output tensor
    auto d_tensor = torch::empty_like(c_tensor);

    // Split tensors into groups based on expert_offsets
    std::vector<torch::Tensor> a_tensors, b_tensors, scale_tensors, c_tensors, d_tensors;

    auto offsets = expert_offsets.cpu().data_ptr<int64_t>();
    int64_t K = a_tensor.size(1);

    for (int i = 0; i < num_groups; i++) {
        // Extract the appropriate slices for each group
        int64_t M_offset = offsets[i];
        int64_t M = (i < num_groups - 1) ? (offsets[i+1] - offsets[i]) : (a_tensor.size(0) - offsets[i]);

        a_tensors.push_back(a_tensor.slice(0, M_offset, M_offset + M));
        b_tensors.push_back(b_tensor);  // Each group uses the whole B tensor
        scale_tensors.push_back(scale_tensor);  // Each group uses the whole scale tensor
        c_tensors.push_back(c_tensor.slice(0, M_offset, M_offset + M));
        d_tensors.push_back(d_tensor.slice(0, M_offset, M_offset + M));
    }

    // Preprocess int4 tensors
    std::vector<torch::Tensor> processed_b_tensors = preprocessInt4Tensors(b_tensors);

    // Preprocess scale tensors
    std::vector<torch::Tensor> packed_scale_tensors = preprocessScaleTensors(scale_tensors);

    // Track if we allocate problem_sizes ourselves
    bool allocated_problem_sizes = false;
    typename ProblemShape::UnderlyingProblemShape* device_problem_sizes = problem_sizes;

    // Collect problem dimensions if not provided
    if (!problem_sizes) {
        std::vector<int> m_values(num_groups), n_values(num_groups), k_values(num_groups);
        std::vector<typename ProblemShape::UnderlyingProblemShape> host_problem_sizes;

        for (int i = 0; i < num_groups; ++i) {
            const auto& a = a_tensors[i];
            const auto& b = b_tensors[i];

            int64_t M = a.size(0);
            int64_t K = a.size(1);
            int64_t N = b.size(0);

            // Verify that K is consistent with packed int4 format in B
            TORCH_CHECK(b.size(1) * 2 == K,
                "B tensor has inconsistent dimensions for int4 format. "
                "B.size(1) should be A.size(1)/2 because each byte stores two int4 values.");

            m_values[i] = static_cast<int>(M);
            n_values[i] = static_cast<int>(N);
            k_values[i] = static_cast<int>(K);
        }

        // Create problem sizes
        host_problem_sizes = createProblemSizes(m_values, n_values, k_values);

        // Allocate device memory for problem sizes and copy
        hipMalloc(&device_problem_sizes, num_groups * sizeof(typename ProblemShape::UnderlyingProblemShape));
        hipMemcpy(
            device_problem_sizes,
            host_problem_sizes.data(),
            num_groups * sizeof(typename ProblemShape::UnderlyingProblemShape),
            hipMemcpyHostToDevice
        );
        allocated_problem_sizes = true;
    } else {
        // We're using provided problem_sizes
        device_problem_sizes = problem_sizes;
    }

    // Create strides for tensors if not provided
    std::vector<StrideA> local_stride_a;
    std::vector<StrideB> local_stride_b;
    std::vector<StrideC> local_stride_c;
    std::vector<StrideD> local_stride_d;
    std::vector<StrideS> local_stride_s;

    StrideA* stride_a_ptr = (StrideA*) a_strides;
    StrideB* stride_b_ptr = (StrideB*) b_strides;
    StrideC* stride_c_ptr = (StrideC*) c_strides;

    if (!a_strides) {
        local_stride_a = createStrides<StrideA>(a_tensors);
        stride_a_ptr = local_stride_a.data();
    }

    if (!b_strides) {
        local_stride_b = createStrides<StrideB>(processed_b_tensors);
        stride_b_ptr = local_stride_b.data();
    }

    if (!c_strides) {
        local_stride_c = createStrides<StrideC>(c_tensors);
        stride_c_ptr = local_stride_c.data();
    }

    // Always create D strides
    local_stride_d = createStrides<StrideD>(d_tensors);

    // Create scale strides
    local_stride_s = createStrides<StrideS>(packed_scale_tensors);

    // Determine chunk_size if not provided
    if (chunk_size <= 0) {
        // Default to K / number of scales in the scale tensor
        int64_t K = a_tensor.size(1);
        int64_t scale_cols = scale_tensor.size(1);
        chunk_size = static_cast<int>(K / scale_cols);
        TORCH_CHECK(chunk_size > 0, "Cannot determine chunk size. Please provide it explicitly.");
    }

    // Get device pointers
    auto a_ptrs = getDevicePtrs<MmaType>(a_tensors);
    auto b_ptrs = getDevicePtrs<QuantType>(processed_b_tensors);
    auto scale_ptrs = getDevicePtrs<ElementScalePacked>(packed_scale_tensors);
    auto c_ptrs = getDevicePtrs<ElementC>(c_tensors);
    auto d_ptrs = getMutableDevicePtrs<ElementD>(d_tensors);

    // Create parameters structure
    Int4Fp8GemmParams params;
    params.num_groups = num_groups;
    params.problem_sizes = device_problem_sizes;
    params.a_ptrs = a_ptrs.data();
    params.b_ptrs = b_ptrs.data();
    params.scale_ptrs = scale_ptrs.data();
    params.c_ptrs = c_ptrs.data();
    params.d_ptrs = d_ptrs.data();
    params.stride_a = stride_a_ptr;
    params.stride_b = stride_b_ptr;
    params.stride_c = stride_c_ptr;
    params.stride_d = local_stride_d.data();
    params.stride_s = local_stride_s.data();

    // Allocate and set alpha and beta values
    ElementAccumulator* alpha_ptr = nullptr;
    ElementAccumulator* beta_ptr = nullptr;
    hipMalloc(&alpha_ptr, sizeof(ElementAccumulator));
    hipMalloc(&beta_ptr, sizeof(ElementAccumulator));
    hipMemcpy(alpha_ptr, &alpha, sizeof(ElementAccumulator), hipMemcpyHostToDevice);
    hipMemcpy(beta_ptr, &beta, sizeof(ElementAccumulator), hipMemcpyHostToDevice);

    params.alpha = alpha_ptr;
    params.beta = beta_ptr;
    params.chunk_size = chunk_size;
    params.workspace = nullptr;  // Allocated inside runInt4Fp8GroupedGemm

    // Run the GEMM
    hipError_t status = runInt4Fp8GroupedGemm(params);

    // Free allocated resources
    if (alpha_ptr) hipFree(alpha_ptr);
    if (beta_ptr) hipFree(beta_ptr);

    // Free problem sizes if we allocated them
    if (allocated_problem_sizes && device_problem_sizes) {
        hipFree(device_problem_sizes);
    }

    if (status != hipSuccess) {
        TORCH_CHECK(false, "int4_fp8_grouped_gemm failed with error: ", hipGetErrorString(status));
    }

    return d_tensor;
}